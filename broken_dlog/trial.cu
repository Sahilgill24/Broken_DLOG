#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <cstdlib>

// kernel to compute our value. 
// this is called from the CPU, but runs on the GPU.
__global__ void bs_kernel(int M, int p, const int *lookup_table, int *j, int g) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < M) {
        lookup_table[j] = idx % p;
        j = (j * g) % p;
        // would this work parallely ? , not sure
        // use the pollard rho as a reference 
        // but write a basic schematic
    }
}

__global__ void gs_kernel(int *a,int g_inv, const int *lookup_table, int M, int p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < M) {
        auto it = std::find(lookup_table.begin(), lookup_table.end(), a);
        if (it != lookup_table.end()) {
            int j = std::distance(lookup_table.begin(), it);
            // now we have the index of a in the lookup table
            // we can compute the value of j
            a = (a * g_inv) % p;
            printf("Found a at index %d, j = %d\n", idx, j);
            return j*m + lookup_table[it - lookup_table.begin()];
            
        } else {
            printf("a not found in lookup table\n");
        }

    }
}





int main() {
    const int M = 1000;
    const int p = 1000000007;
    const int g = 3;
    // this is not g_inv, this is g^m 's inverse mod p
    const int gm_inv = 333333336; 
    const int a = 5;
    int j = 1;
    std::vector<int> lookup_table(M);
    int *d_lookup_table;
    hipMalloc(&d_lookup_table, M * sizeof(int));
    bs_kernel<<<(M + 255) / 256, 256>>>(M, p, d_lookup_table, &j, g);
    hipMemcpy(lookup_table.data(), d_lookup_table, M * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup_table);
    return 0;
}